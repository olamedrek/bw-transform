
#include <hip/hip_runtime.h>
#include <cstdio>

extern "C" {
__global__
/* Result:
	[pos] - target positions in blocks only
	[zerosInBlocks] - for each block number of zeros in that block */
void computeLocalPositions(int* in, int n, int* pos, int k, int* zerosInBlocks) {
	int thid = blockDim.x * blockIdx.x + threadIdx.x;
	int id = threadIdx.x;

	__shared__ int sh[2048];

	int realShSize = 
		blockIdx.x == gridDim.x - 1 ? 
		n - blockIdx.x * 2048 : 2048;

	if(id >= realShSize) return;

	int bit = k < 32 ? 
		(in[3 * thid] >> k) & 1 : 
		(in[3 * thid + 1] >> (k - 32)) & 1;

	sh[id] = bit;

	for(int offset = 1; offset < 2048; offset *= 2) {
		__syncthreads();

		int tmp = 0;
		if(id >= offset)
			tmp = sh[id - offset];

		__syncthreads();

		sh[id] += tmp;
	}

	__syncthreads();

	int zeros = realShSize - sh[realShSize - 1];

	pos[thid] = bit ? 
		zeros + sh[id] - 1 : id - sh[id];

	if(id == 0)
		zerosInBlocks[blockIdx.x] = zeros;
}
}

extern "C" {
__global__
/* 
Parameters:
	[zerosPref] - prefix sum of array [zerosInBlock] (filled by function radixsort)
Result:
	[pos] - global target positions */
void computeGlobalPositions(int* in, int n, int* pos, int k, int* zerosPref) {
	int thid = blockDim.x * blockIdx.x + threadIdx.x;

	if(thid >= n) return;

	int bit = k < 32 ? 
		(in[3 * thid] >> k) & 1 : 
		(in[3 * thid + 1] >> (k - 32)) & 1;

	int elementsBefore = blockIdx.x * 2048;

	int zerosBefore = 
		blockIdx.x == 0 ? 
		0 : zerosPref[blockIdx.x - 1];

	int zerosAfter = zerosPref[gridDim.x - 1] - zerosPref[blockIdx.x];
	
	if(bit == 0) {
		pos[thid] += zerosBefore;
	} else {
		pos[thid] += zerosAfter + elementsBefore;
	}
}
}

extern "C" {
__global__
/* This function rewrites elements from array [in] to array [out] at their correct positions in order given in array [pos] */ 
void permute(int* in, int n, int* out, int* pos) {
	int thid = blockDim.x * blockIdx.x + threadIdx.x;

	if(thid >= n) return;

	out[3 * pos[thid]] = in[3 * thid];
	out[3 * pos[thid] + 1] = in[3 * thid + 1];
	out[3 * pos[thid] + 2] = in[3 * thid + 2];
}
}
